#include "hip/hip_runtime.h"
#include "kernel.h"
#include <math.h>
#include <cmath>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
//divide
#define THREADS_PER_SAMPLE 16
#define SAMPLES_PER_THREAD 1
#define SAMPLING_FREQ 44100
//#define SIMPLE 0
#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)
float *dev_buffer, *dev_tmp_buffer;
float slideTime;
int numSamples, numSinusoids, numVoices;
void printArraywNewLines(int n, float *a, bool abridged) {
    printf("    [ ");
    for (int i = 0; i < n; i++) {
        if (abridged && i + 2 == 15 && n > 16) {
            i = n - 2;
            printf("... ");
        }
        printf("%3f\n", a[i]);
    }
    printf("]\n");
}
void printArraywNewLines(int n, float2 *a, bool abridged) {
    printf("    [ ");
    for (int i = 0; i < n; i++) {
        if (abridged && i + 2 == 15 && n > 16) {
            i = n - 2;
            printf("... ");
        }
        printf("%3f, ", a[i].x);
        printf("%3f\n", a[i].y);
    }
    printf("]\n");
}
void printArray(int n, float2 *a, bool abridged) {
    printf("    [ ");
    for (int i = 0; i < n; i++) {
        if (abridged && i + 2 == 15 && n > 16) {
            i = n - 2;
            printf("... ");
        }
        printf("{%3f, ", a[i].x);
        printf("%3f},", a[i].y);
    }
    printf("]\n");
}

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

void Additive::alloc_engine(float2* &h_freq_gains, float* &h_angles, float* &h_v_gains, float* &h_tmp_buffer,
 float* &h_buffer,float* &h_adsr, int num_samples, int num_voices, int num_harms){
	 hipHostAlloc((void**)&h_freq_gains, sizeof(float2)*num_harms * num_voices, hipHostMallocMapped);
          checkCUDAError("h_freq_gains get Device Pointer", __LINE__);
          hipHostAlloc((void**)&h_angles, sizeof(float)*num_harms*num_voices, hipHostMallocMapped);
          checkCUDAError("h_angles get Device Pointer", __LINE__);
          hipHostAlloc((void**)&h_v_gains, sizeof(float)*num_voices, hipHostMallocMapped);
          checkCUDAError("h_v_gains get Device Pointer", __LINE__);
          hipHostAlloc((void**)&h_tmp_buffer, sizeof(float)*num_samples, hipHostMallocMapped);
          checkCUDAError("h_tmp_buffer get Device Pointer", __LINE__);
          hipHostAlloc((void**)&h_buffer, sizeof(float)*num_samples, hipHostMallocMapped);
          checkCUDAError("h_buffer get Device Pointer", __LINE__);
          hipHostAlloc((void**)&h_adsr, sizeof(float)*num_voices * num_samples,hipHostMallocMapped);
          checkCUDAError("h_Adsrs get Device Pointer", __LINE__);
	
}

void Additive::realloc_engine(float* &h_tmp_buffer,float* &h_buffer, int prev_num_samples, int num_samples){
          float tmp_buffer[num_samples];
		  float buffer[num_samples];
          hipMemcpy(tmp_buffer, h_tmp_buffer, sizeof(float)*prev_num_samples, hipMemcpyHostToHost);
		  checkCUDAError("memcpy realloc firs tmp_buffer", __LINE__);
		  hipMemcpy(buffer, h_buffer, sizeof(float)*prev_num_samples, hipMemcpyHostToHost); 
		  checkCUDAError("memcpy realloc first buffer", __LINE__);
		  hipFree(h_tmp_buffer);
		  hipFree(h_buffer);
          hipHostAlloc((void**)&h_tmp_buffer, sizeof(float)*num_samples, hipHostMallocMapped);
          checkCUDAError("h_tmp_buffer get Device Pointer", __LINE__);
          hipHostAlloc((void**)&h_buffer, sizeof(float)*num_samples, hipHostMallocMapped);
          checkCUDAError("h_buffer get Device Pointer", __LINE__);
          
		  hipMemcpy(h_tmp_buffer,tmp_buffer, sizeof(float)*prev_num_samples, hipMemcpyHostToHost);
		      checkCUDAError("memcpy realloc tmp_buffer", __LINE__);
		  hipMemcpy(h_buffer,buffer,  sizeof(float)*prev_num_samples, hipMemcpyHostToHost); 
		      checkCUDAError("memcpy reallox buffer", __LINE__);

}
__global__ void my_vh_kernel(float *outBuffer, float2 *freq_gains, float *vgains, float* adsr, float angle, int numSamples, int numSinusoids, int numVoices)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numSamples) {
		// samples sine wave in discrete steps
		angle = angle + 2.f * M_PI * idx / 44100.f;
		
		float buff_val = 0.f;
		int numHarmonics = numSinusoids / numVoices;

		for (int i = 0; i < numVoices; i++) {
			for (int j = 0; j < numHarmonics; j++) {
				float gain = vgains[i] * adsr[i * idx] * freq_gains[i*numHarmonics + j].y;
				buff_val += gain * (__sinf(angle * freq_gains[i*numHarmonics + j].x) + 0.5f);
				//printf("idx %d buff val: %f\n", idx, buff_val);
			}
		}

		// buffer to be sent to DAC
		outBuffer[idx] = buff_val;
	}
}

void Additive::my_v_compute(float *buffer, float angle, 
	float* h_buffer, float* h_v_gains,
	 float2* h_freq_gains, float* h_adsr, int numSamples,
		int numSinusoids, int numVoices) 
	{
		//static int count = 0;
		//std::cout << "frequency" << std::endl;
		//printArray(1, h_freq_gains, 0);
		int threadsPerBlock = numSamples;
		int blocksPerGrid = (numSamples + threadsPerBlock - 1) / threadsPerBlock;
		float *dev_buffer, *dev_v_gains, *dev_adsr;
		float2* dev_freq_gains;
		hipHostGetDevicePointer((void**)&dev_freq_gains, (void*)h_freq_gains, 0);
		checkCUDAError("dev_freq_gains get Device Pointer", __LINE__);

		//hipHostGetDevicePointer((void**)&dev_angles, (void*)h_angles, 0);
		hipHostGetDevicePointer((void**)&dev_v_gains, (void*)h_v_gains, 0);
		checkCUDAError("dev_v_gains get Device Pointer", __LINE__);
	
		hipHostGetDevicePointer((void**)&dev_buffer, (void*)h_buffer, 0);
		checkCUDAError("dev_buffer get Device Pointer", __LINE__);

		hipHostGetDevicePointer((void**)&dev_adsr, (void*)h_adsr, 0);
		checkCUDAError("dev_freq_gains get Device Pointer", __LINE__);

		
		my_vh_kernel <<< blocksPerGrid, threadsPerBlock >>> (dev_buffer, dev_freq_gains, dev_v_gains, dev_adsr,
										angle, numSamples, numSinusoids, numVoices);
		checkCUDAError("vhkernel error", __LINE__);
		hipStreamSynchronize(NULL);
				//std::cout << "reyeet"<<std::endl;
		#ifdef KERNELDEBUG
           float *debug_arr1;
          hipMallocManaged((void**)&debug_arr1, sizeof(float) * numSamples, hipMemAttachHost);
          checkCUDAError("Error debugging output after cufftshift (malloc)", __LINE__);
          hipMemcpy(debug_arr1,dev_buffer, sizeof(float) *numSamples,hipMemcpyDeviceToHost);
          checkCUDAError("Error debugging output after cufftshift (memcpy)", __LINE__);
          printf("out\n");
          printArraywNewLines(numSamples, debug_arr1, 0);
          hipFree(debug_arr1);
		#endif
    //std::cout << "yeet" << std::endl;
		// updates the buffer with dev_buffer computed in GPU
		hipMemcpy(buffer, dev_buffer, numSamples * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAError("memcpy error", __LINE__);
		#ifdef DEBUGCOPYM
		std::cout << "kernel" << std::endl;
		  float *debug_arr1;
          hipMallocManaged((void**)&debug_arr1, sizeof(float) * numSamples, hipMemAttachHost);
          checkCUDAError("Error debugging output after cufftshift (malloc)", __LINE__);
         memcpy(debug_arr1,buffer, sizeof(float) *numSamples);
          checkCUDAError("Error debugging output after cufftshift (memcpy)", __LINE__);
          printf("out\n");
          printArraywNewLines(numSamples, debug_arr1, 1);
          hipFree(debug_arr1);
		#endif
		
		
}


__device__ float ramp_kern(float currentTime, float slideTime, float f0, float f1){
	float integral;
	if (currentTime < slideTime) {
		float k = (f1-f0) / slideTime;
		integral = currentTime * (f0 + k * currentTime / 2.0f);
	} else {
		integral = f0 * slideTime + (f1 - f0) * slideTime / 2.0f;
		integral += (currentTime - slideTime) * f1;
	}
	return integral * 2.0f * M_PI;
}

#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )

__global__ void sin_kernel_fast(float2* freq_gains, float* buffer, 
								float* angles, int numThreadsPerBlock, int numSinusoids,
								float time, int numSamples) 
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numSamples * THREADS_PER_SAMPLE) {
		//determine how many sineWaves are to be computed in each thread based on how many threads it takes to compute a sample
		int maxSinePerBlock = (numSinusoids + THREADS_PER_SAMPLE - 1) / THREADS_PER_SAMPLE;
		int sinBlock = idx / numThreadsPerBlock;
		int sampleIdx = idx - sinBlock * numThreadsPerBlock; // modulo function but GPUs are trash at modulo so don't use it
		float val[SAMPLES_PER_THREAD];
		for (int j = 0; j < SAMPLES_PER_THREAD; j++) {
			val[j] = 0.0f;
		}
	    float gain, freq0, angle, angleStart;
	    int firstSine = sinBlock * maxSinePerBlock;
		int lastSine = imin(numSinusoids, firstSine + maxSinePerBlock);
		//compute samples for maxSinePerBlock
		for (int i = firstSine; i < lastSine; i++) {
			angleStart = angles[i]; 
			freq0 = freq_gains[i].x;
			gain = freq_gains[i].y;
			for (int j = 0; j < SAMPLES_PER_THREAD; j++) {
			  angle = angleStart + time + (sampleIdx*SAMPLES_PER_THREAD+j) / SAMPLING_FREQ;
				val[j] += __sinf(angle * freq0) * gain / numSinusoids;
			}
			angles[i] = fmod(angle, 44100.f);
		}
		for (int i = 0; i < SAMPLES_PER_THREAD; i++) {
			buffer[idx * SAMPLES_PER_THREAD + i] = val[i];
		}

	}

}


__global__ void sum_blocks(float* tmp_buffer, float* buffer, int numSamples) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numSamples) {
		float sum = 0;
		for (int i = 0; i < THREADS_PER_SAMPLE; i++) {
			sum += tmp_buffer[idx + i * numSamples];
		}
		buffer[idx] = sum;
	}

}



void Additive::compute_sinusoid_hybrid(float* samples, float2* h_freq_gains, float* h_angles, float *h_v_gains, float* h_tmp_buffer, float* h_buffer, int numSinusoids, float time, float numSamples){
	int threadsPerBlock = 256; 
	int numThreadsPerBlock = numSamples / SAMPLES_PER_THREAD;
	int numThreads = THREADS_PER_SAMPLE * numThreadsPerBlock;
	int blocksPerGrid = (numThreads + threadsPerBlock - 1) / threadsPerBlock;
	float2* dev_freqs_gains;
	float* dev_buffer,* dev_tmp_buffer, *dev_angles, *dev_v_gains;
	hipHostGetDevicePointer((void**)&dev_freqs_gains, (void*)h_freq_gains, 0);
	checkCUDAError("dev_freq_gains get Device Pointer", __LINE__);
	hipHostGetDevicePointer((void**)&dev_angles, (void*)h_angles, 0);
	checkCUDAError("dev_angles get Device Pointer", __LINE__);
	hipHostGetDevicePointer((void**)&dev_v_gains, (void*)h_v_gains, 0);
	checkCUDAError("dev_v_gains get Device Pointer", __LINE__);
	hipHostGetDevicePointer((void**)&dev_tmp_buffer, (void*)h_tmp_buffer, 0);
	checkCUDAError("dev_tmp_buffer get Device Pointer", __LINE__);
	hipHostGetDevicePointer((void**)&dev_buffer, (void*)h_buffer, 0);
	checkCUDAError("dev_buffer get Device Pointer", __LINE__);


	sin_kernel_fast <<<blocksPerGrid, threadsPerBlock >>>(dev_freqs_gains, dev_tmp_buffer, dev_angles, numThreadsPerBlock, numSinusoids, time, numSamples);
	//checkCUDAErrorWithLine("sin_kernel_fast failed");
	blocksPerGrid = (numSamples + threadsPerBlock - 1) / threadsPerBlock;
	sum_blocks <<<blocksPerGrid, threadsPerBlock >> >(dev_tmp_buffer, dev_buffer, numSamples);
	//checkCUDAErrorWithLine("sum_blocks failed");
	hipMemcpy(samples, dev_buffer, numSamples * sizeof(float), hipMemcpyDeviceToHost);
}

