#include "hip/hip_runtime.h"
#include "kernel.h"
#include <math.h>
#include <cmath>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "kernel.h"
#include <iostream>
#define M_PI 3.1415926535897931
//divide
#define THREADS_PER_SAMPLE 16
#define SAMPLES_PER_THREAD 1
#define SAMPLING_FREQ 44100
//#define SIMPLE 0
#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)
float *dev_buffer, *dev_tmp_buffer;
float slideTime;
int numSamples, numSinusoids, numVoices;

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
__global__ void my_vh_kernel(float *outBuffer, float2 *freq_gains, float *vgains, float angle, int numSamples, int numSinusoids, int numVoices)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numSamples) {
		// samples sine wave in discrete steps
		angle = angle + 2.f * M_PI * idx / 44100.f;
		float buff_val = 0.f;
		int numHarmonics = numSinusoids / numVoices;

		for (int i = 0; i < numVoices; i++) {
			for (int j = 0; j < numHarmonics; j++) {
				float gain = vgains[i] * freq_gains[i*numHarmonics + j].y;
				buff_val += gain * (__sinf(angle * freq_gains[i*numHarmonics + j].x) + 0.5f);
			}
		}

		// buffer to be sent to DAC
		outBuffer[idx] = buff_val;
	}
}

void Additive::my_v_compute(float *buffer, float angle, 
		float* h_buffer, float* h_v_gains,
		float2* h_freq_gains, int numSamples,
		int numSinusoids, int numVoices) 
	{
		int threadsPerBlock = 256;
		int blocksPerGrid = (numSamples + threadsPerBlock - 1) / threadsPerBlock;
		float *dev_buffer, *dev_v_gains;
		float2* dev_freq_gains;

		hipHostGetDevicePointer((void**)&dev_freq_gains, (void*)h_freq_gains, 0);
		checkCUDAError("dev_freq_gains get Device Pointer", __LINE__);

		//hipHostGetDevicePointer((void**)&dev_angles, (void*)h_angles, 0);
		hipHostGetDevicePointer((void**)&dev_v_gains, (void*)h_v_gains, 0);
		checkCUDAError("dev_v_gains get Device Pointer", __LINE__);
	
		hipHostGetDevicePointer((void**)&dev_buffer, (void*)h_buffer, 0);
		checkCUDAError("dev_buffer get Device Pointer", __LINE__);

		my_vh_kernel <<< blocksPerGrid, threadsPerBlock >>> (dev_buffer, dev_freq_gains, dev_v_gains,
										angle, numSamples, numSinusoids, numVoices);
										
		// updates the buffer with dev_buffer computed in GPU
		hipMemcpy(buffer, dev_buffer, numSamples * sizeof(float), hipMemcpyDeviceToHost);
}


__device__ float ramp_kern(float currentTime, float slideTime, float f0, float f1){
	float integral;
	if (currentTime < slideTime) {
		float k = (f1-f0) / slideTime;
		integral = currentTime * (f0 + k * currentTime / 2.0f);
	} else {
		integral = f0 * slideTime + (f1 - f0) * slideTime / 2.0f;
		integral += (currentTime - slideTime) * f1;
	}
	return integral * 2.0f * M_PI;
}

#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )

__global__ void sin_kernel_fast(float2* freq_gains, float* buffer, 
								float* angles, int numThreadsPerBlock, int numSinusoids,
								float time, int numSamples) 
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numSamples * THREADS_PER_SAMPLE) {
		//determine how many sineWaves are to be computed in each thread based on how many threads it takes to compute a sample
		int maxSinePerBlock = (numSinusoids + THREADS_PER_SAMPLE - 1) / THREADS_PER_SAMPLE;
		int sinBlock = idx / numThreadsPerBlock;
		int sampleIdx = idx - sinBlock * numThreadsPerBlock; // modulo function but GPUs are trash at modulo so don't use it
		float val[SAMPLES_PER_THREAD];
		for (int j = 0; j < SAMPLES_PER_THREAD; j++) {
			val[j] = 0.0f;
		}
	    float gain, freq0, angle, angleStart;
	    int firstSine = sinBlock * maxSinePerBlock;
		int lastSine = imin(numSinusoids, firstSine + maxSinePerBlock);
		//compute samples for maxSinePerBlock
		for (int i = firstSine; i < lastSine; i++) {
			angleStart = angles[i]; 
			freq0 = freq_gains[i].x;
			gain = freq_gains[i].y;
			for (int j = 0; j < SAMPLES_PER_THREAD; j++) {
			  angle = angleStart + time + (sampleIdx*SAMPLES_PER_THREAD+j) / SAMPLING_FREQ;
				val[j] += __sinf(angle * freq0) * gain / numSinusoids;
			}
			angles[i] = angleStart + angle;
		}
		for (int i = 0; i < SAMPLES_PER_THREAD; i++) {
			buffer[idx * SAMPLES_PER_THREAD + i] = val[i];
		}

	}

}


__global__ void sum_blocks(float* tmp_buffer, float* buffer, int numSamples) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numSamples) {
		float sum = 0;
		for (int i = 0; i < THREADS_PER_SAMPLE; i++) {
			sum += tmp_buffer[idx + i * numSamples];
		}
		buffer[idx] = sum;
	}

}



	void Additive::compute_sinusoid_hybrid(float* samples, float2* h_freq_gains, float* h_angles, float *h_v_gains, float* h_tmp_buffer, float* h_buffer, int numSinusoids, float time, float numSamples){
int threadsPerBlock = 256; 
int numThreadsPerBlock = numSamples / SAMPLES_PER_THREAD;
int numThreads = THREADS_PER_SAMPLE * numThreadsPerBlock;
int blocksPerGrid = (numThreads + threadsPerBlock - 1) / threadsPerBlock;
float2* dev_freqs_gains;
float* dev_buffer,* dev_tmp_buffer, *dev_angles, *dev_v_gains;
hipHostGetDevicePointer((void**)&dev_freqs_gains, (void*)h_freq_gains, 0);
checkCUDAError("dev_freq_gains get Device Pointer", __LINE__);
hipHostGetDevicePointer((void**)&dev_angles, (void*)h_angles, 0);
checkCUDAError("dev_angles get Device Pointer", __LINE__);
hipHostGetDevicePointer((void**)&dev_v_gains, (void*)h_v_gains, 0);
checkCUDAError("dev_v_gains get Device Pointer", __LINE__);
hipHostGetDevicePointer((void**)&dev_tmp_buffer, (void*)h_tmp_buffer, 0);
checkCUDAError("dev_tmp_buffer get Device Pointer", __LINE__);
hipHostGetDevicePointer((void**)&dev_buffer, (void*)h_buffer, 0);
checkCUDAError("dev_buffer get Device Pointer", __LINE__);
sin_kernel_fast <<<blocksPerGrid, threadsPerBlock >>>(dev_freqs_gains, dev_tmp_buffer, dev_angles, numThreadsPerBlock, numSinusoids, time, numSamples);
//checkCUDAErrorWithLine("sin_kernel_fast failed");
blocksPerGrid = (numSamples + threadsPerBlock - 1) / threadsPerBlock;
sum_blocks <<<blocksPerGrid, threadsPerBlock >> >(dev_tmp_buffer, dev_buffer, numSamples);
//checkCUDAErrorWithLine("sum_blocks failed");
hipMemcpy(samples, dev_buffer, numSamples * sizeof(float), hipMemcpyDeviceToHost);
}

