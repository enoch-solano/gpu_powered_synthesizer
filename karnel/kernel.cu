#include "hip/hip_runtime.h"
#include "kernel.h"
#include <math.h>
#include <cmath>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "kernel.h"
#define M_PI 3.1415926535897931
//divide
#define THREADS_PER_SAMPLE 16
#define SAMPLES_PER_THREAD 1
#define SAMPLING_FREQ 44100
//#define SIMPLE 0
#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)
float* dev_freqs, *dev_gains, *dev_vgains, *dev_buffer, *dev_tmp_buffer, *dev_target, *dev_angle;
float slideTime;
int numSamples, numSinusoids, numVoices;

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//************************************* my synth with voice/harmonics *************************************//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void my_vh_kernel(float *outBuffer, float *freqs, float *gains, float *vgains,
								float angle, int numSamples, int numSinusoids, int numVoices)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numSamples) {
		// samples sine wave in discrete steps
		angle = angle + 2.f * M_PI * idx / 44100.f;
		float buff_val = 0.f;
		int numHarmonics = numSinusoids / numVoices;

		for (int i = 0; i < numVoices; i++) {
			for (int j = 0; j < numHarmonics; j++) {
				buff_val += vgains[i] * gains[i*numHarmonics + j] * __sinf(angle * freqs[i*numHarmonics + j]);
			}
		}

		// buffer to be sent to DAC
		outBuffer[idx] = buff_val;
	}
}

void Additive::my_v_compute(float *buffer, float angle) {
	int threadsPerBlock = 256;
	int blocksPerGrid = (numSamples + threadsPerBlock - 1) / threadsPerBlock;

	my_vh_kernel <<< blocksPerGrid, threadsPerBlock >>> (dev_buffer, dev_freqs, dev_gains, dev_vgains,
															angle, numSamples, numSinusoids, numVoices);

	// updates the buffer with dev_buffer computed in GPU
	hipMemcpy(buffer, dev_buffer, numSamples * sizeof(float), hipMemcpyDeviceToHost);
}

void Additive::initVSynth(int numSample, const v_udata& v_user_data) {
	// initializes global variables with appropriate values
	numSamples = numSample;
	numSinusoids = NUM_VOICES * NUM_HARMS;
	numVoices = NUM_VOICES;

	// allocates memory in GPU
	hipMalloc((void**)&dev_freqs, numSinusoids * sizeof(float));
	checkCUDAErrorWithLine("dev_freqs malloc failed");
	hipMalloc((void**)&dev_gains, numSinusoids * sizeof(float));
	checkCUDAErrorWithLine("dev_gains malloc failed");
	hipMalloc((void**)&dev_vgains, numVoices * sizeof(float));
	checkCUDAErrorWithLine("dev_vgains malloc failed");
	hipMalloc((void**)&dev_buffer, numSamples * sizeof(float));
	checkCUDAErrorWithLine("dev_buffer malloc failed");

	// copy memory from CPU to GPU
	hipMemcpy(dev_freqs, v_user_data.freqs, numSinusoids * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("dev_freqs memcpy failed");
	hipMemcpy(dev_gains, v_user_data.gains, numSinusoids * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("dev_gains memcpy failed");
	hipMemcpy(dev_vgains, v_user_data.v_gains, numVoices * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("dev_vgains memcpy failed");

	hipDeviceSynchronize();
}
void Additive::updateFreqsVSynth(float *freqs) {
	// copy memory from CPU to GPU
	hipMemcpy(dev_freqs, freqs, numSinusoids * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("frequencies memcpy failed");
}
void Additive::updateGainsVSynth(float *gains) {
	// copy memory from CPU to GPU
	hipMemcpy(dev_gains, gains, numSinusoids * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("gains memcpy failed");
}
void Additive::updateVGainsVSynth(float *v_gains) {
	// copy memory from CPU to GPU
	hipMemcpy(dev_vgains, v_gains, numVoices * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("voice gains memcpy failed");
}
void Additive::endVSynth() {
	hipFree(dev_freqs);
	hipFree(dev_gains);
	hipFree(dev_vgains);
	hipFree(dev_buffer);
}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//************************************* my simple (interactive) synth *************************************//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void my_simple_kernel(float *outBuffer, float *freqs, float *gains, 
								 float angle, int numSamples, int numSinusoids)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numSamples) {
		// samples sine wave in discrete steps
		angle = angle + 2.f * M_PI * idx / 44100.f;
		float buff_val = 0.f;

		for (int i = 0; i < 16 /*NUM_SINUSOIDS*/; i++) {
			buff_val += gains[i] * __sinf(angle * freqs[i]);
		}

		// buffer to be sent to DAC
		outBuffer[idx] = buff_val;
	}
}

void Additive::my_simple_compute(float *buffer, float angle)
{
	int threadsPerBlock = 256;
	int blocksPerGrid = (numSamples + threadsPerBlock - 1) / threadsPerBlock;

	my_simple_kernel <<< blocksPerGrid, threadsPerBlock >>> (dev_buffer, dev_freqs, dev_gains,
															 angle, numSamples, numSinusoids);

	// updates the buffer with buffer computed in GPU
	hipMemcpy(buffer, dev_buffer, numSamples * sizeof(float), hipMemcpyDeviceToHost);
}

void Additive::initSimpleSynth(int numSinusoid, int numSample, float *init_freqs, float *init_gains)
{
	// initializes global variables with appropriate values
	numSamples = numSample;
	numSinusoids = numSinusoid;

	// allocates memory in GPU
	hipMalloc((void**)&dev_freqs, numSinusoids * sizeof(float));
	checkCUDAErrorWithLine("dev_freqs malloc failed");
	hipMalloc((void**)&dev_gains, numSinusoids * sizeof(float));
	checkCUDAErrorWithLine("dev_gains malloc failed");
	hipMalloc((void**)&dev_buffer, numSamples * sizeof(float));
	checkCUDAErrorWithLine("dev_buffer malloc failed");

	// copy memory from CPU to GPU
	hipMemcpy(dev_freqs, init_freqs, numSinusoids * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("dev_freqs memcpy failed");
	hipMemcpy(dev_gains, init_gains, numSinusoids * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("dev_gains memcpy failed");

	hipDeviceSynchronize();
}

void Additive::updateFreqsSimpleSynth(float *freqs) {
	// copy memory from CPU to GPU
	hipMemcpy(dev_freqs, freqs, numSinusoids * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("initial frequencies memcpy failed");
}

void Additive::updateGainsSimpleSynth(float *gains) {
	// copy memory from CPU to GPU
	hipMemcpy(dev_gains, gains, numSinusoids * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("dev_gains memcpy failed");
}


void Additive::endSimpleSynth() {
	hipFree(dev_freqs);
	hipFree(dev_buffer);
	hipFree(dev_gains);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////


void Additive::initSynth(int numSinusoid, int numSample, float* host_frequencies) {
	
	numSamples = numSample;
	numSinusoids = numSinusoid;
	hipMalloc((void**)&dev_freqs, numSinusoids * sizeof(float));
	hipMalloc((void**)&dev_buffer, numSamples * sizeof(float));
	hipMemcpy(dev_freqs, host_frequencies, numSinusoids * sizeof(float), hipMemcpyHostToDevice);
	
	hipDeviceSynchronize();
}

void Additive::initSynth_THX(int numSinusoid, int numSample, float* host_start_freq, float* host_end_freq, float* host_angle, float*  host_gains, float slide) {
	numSamples = numSample;
	numSinusoids = numSinusoid;
	slideTime = slide;

	hipMalloc((void**)&dev_freqs, numSinusoids * sizeof(float));
	checkCUDAErrorWithLine("dev_freqs malloc failed");
	hipMemcpy(dev_freqs, host_start_freq, numSinusoids * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("dev_freqs memcpy failed");
	hipMalloc((void**)&dev_buffer, numSamples * sizeof(float));
	checkCUDAErrorWithLine("dev_buffer malloc failed");
	hipMalloc((void**)&dev_tmp_buffer, numSamples *THREADS_PER_SAMPLE* sizeof(float));
	checkCUDAErrorWithLine("dev_tmp_buffer malloc failed");
	hipMalloc((void**)&dev_gains, numSinusoids * sizeof(float));
	checkCUDAErrorWithLine("dev_gains malloc failed");
	hipMemcpy(dev_gains, host_gains, numSinusoids * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("dev_gains memcpy failed");
	hipMalloc((void**)&dev_angle, numSinusoids * sizeof(float));
	checkCUDAErrorWithLine("dev_angle malloc failed");
	//hipMemcpy(dev_angle, host_angle, numSinusoids * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_target, numSinusoids * sizeof(float));
	checkCUDAErrorWithLine("dev_target malloc failed");
	hipMemcpy(dev_target, host_end_freq, numSinusoids * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("target frequencies copy failed");

	hipDeviceSynchronize();
}

void Additive::endSynth_THX() {
	hipFree(dev_freqs);
	hipFree(dev_buffer);
	hipFree(dev_tmp_buffer);
	hipFree(dev_gains);
	hipFree(dev_angle);
	hipFree(dev_target);
}
void Additive::endSynth() {
	hipFree(dev_buffer);
	hipFree(dev_freqs);
}

__global__ void sin_kernel_simple(float *outBuffer, float *frequencies, float angle, int numSamples, int numSinusoids) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numSamples) {
		angle = angle + 2.0f * M_PI * idx / 44100.f;
		float val = 0.0f;
		for (int i = 0; i < numSinusoids; i++) {
			val +=  0.1 * __sinf((angle * frequencies[i]));
		}

		outBuffer[idx] = val;
	}
}

void Additive::compute_sinusoid_gpu_simple(float* buffer, float angle) {
	int threadsPerBlock = 256; 
	int blocksPerGrid = (numSamples + threadsPerBlock - 1) / threadsPerBlock;

	sin_kernel_simple <<< blocksPerGrid, threadsPerBlock >>> (dev_buffer, dev_freqs, angle, numSamples, numSinusoids);
//	sin_kernel_simple <<< 1, 256 >>> (dev_buffer, dev_freqs, angle, numSamples, numSinusoids);
	
	hipMemcpy(buffer, dev_buffer, numSamples * sizeof(float), hipMemcpyDeviceToHost);
}

__device__ float ramp_kern(float currentTime, float slideTime, float f0, float f1){
	float integral;
	if (currentTime < slideTime) {
		float k = (f1-f0) / slideTime;
		integral = currentTime * (f0 + k * currentTime / 2.0f);
	} else {
		integral = f0 * slideTime + (f1 - f0) * slideTime / 2.0f;
		integral += (currentTime - slideTime) * f1;
	}
	return integral * 2.0f * M_PI;
}

#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )

__global__ void sin_kernel_fast(float* buffer, float* frequencies, float* targetFrequencies, 
								float* angles, float* gains, int numThreadsPerBlock, int numSinusoids,
								float time, float slideTime, int numSamples) 
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numSamples * THREADS_PER_SAMPLE) {
		//determine how many sineWaves are to be computed in each thread based on how many threads it takes to compute a sample
		int maxSinePerBlock = (numSinusoids + THREADS_PER_SAMPLE - 1) / THREADS_PER_SAMPLE;
		int sinBlock = idx / numThreadsPerBlock;
		int sampleIdx = idx - sinBlock * numThreadsPerBlock; // modulo function but GPUs are trash at modulo so don't use it
		float val[SAMPLES_PER_THREAD];
		for (int j = 0; j < SAMPLES_PER_THREAD; j++) {
			val[j] = 0.0f;
		}
	    float gain, freq0, freq1, angle, angleStart;
	    int firstSine = sinBlock * maxSinePerBlock;
		int lastSine = imin(numSinusoids, firstSine + maxSinePerBlock);
		//compute samples for maxSinePerBlock
		for (int i = firstSine; i < lastSine; i++) {
			angleStart = 0; 
			freq0 = frequencies[i];
			freq1 = targetFrequencies[i];
			gain = gains[i];
			for (int j = 0; j < SAMPLES_PER_THREAD; j++) {
				angle = ramp_kern(time + (sampleIdx * SAMPLES_PER_THREAD + j) / SAMPLING_FREQ, slideTime, freq0, freq1);
				val[j] += __sinf(angleStart + angle) * gain / numSinusoids;
			}
		}
		for (int i = 0; i < SAMPLES_PER_THREAD; i++) {
			buffer[idx * SAMPLES_PER_THREAD + i] = val[i];
		}

	}

}


__global__ void sum_blocks(float* tmp_buffer, float* buffer, int numSamples) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < numSamples) {
		float sum = 0;
		for (int i = 0; i < THREADS_PER_SAMPLE; i++) {
			sum += tmp_buffer[idx + i * numSamples];
		}
		buffer[idx] = sum;
	}

}

void Additive::compute_sinusoid_hybrid(float* buffer, float * time){
	int threadsPerBlock = 256; 
	int numThreadsPerBlock = numSamples / SAMPLES_PER_THREAD;
	int numThreads = THREADS_PER_SAMPLE * numThreadsPerBlock;
	int blocksPerGrid = (numThreads + threadsPerBlock - 1) / threadsPerBlock;

	
	sin_kernel_fast <<<blocksPerGrid, threadsPerBlock >>>(dev_tmp_buffer, dev_freqs, dev_target, dev_angle, dev_gains, numThreadsPerBlock, numSinusoids, *time, slideTime, numSamples);
	//checkCUDAErrorWithLine("sin_kernel_fast failed");
	blocksPerGrid = (numSamples + threadsPerBlock - 1) / threadsPerBlock;
	sum_blocks <<<blocksPerGrid, threadsPerBlock >> >(dev_tmp_buffer, dev_buffer, numSamples);
	//checkCUDAErrorWithLine("sum_blocks failed");
	hipMemcpy(buffer, dev_buffer, numSamples * sizeof(float), hipMemcpyDeviceToHost);
}
